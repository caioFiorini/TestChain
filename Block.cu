#include "hip/hip_runtime.h"
#include "Block.cuh"
#include "sha256.h"

Block::Block(uint32_t nIndexIn, const string &sDataIn) : _nIndex(nIndexIn), _sData(sDataIn)
{
    _nNonce = 0;
    _tTime = time(nullptr);

    sHash = _CalculateHash();
}

__global__ void mineblock(uint32_t nDifficulty, uint32_t* _nNonce, char* sHash)
{
    char cstr[nDifficulty + 1];
    for (uint32_t i = 0; i < nDifficulty; ++i)
    {
        cstr[i] = '0';
    }
    cstr[nDifficulty] = '\0';

    string str(cstr);

    do
    {
        _nNonce++; // variável da CPU.
        sHash = _CalculateHash(); //variável compartilhada com a CPU.
    } while (sHash.substr(0, nDifficulty) != str);
}

//posso paralelizar essa parte do código.
void Block::MineBlock(uint32_t nDifficulty)
{
    hipMallocManaged(&_nNonce, sizeof(uint32_t));
    // nDifficulty + 1 -> seria adicionar espaço para o /0
    hipMallocManaged(&sHash, (nDifficulty+1) * sizeof(char));

    *_nNonce = 0;
    for(uint32_t i = 0; i < nDifficulty; i++){
        sHash[i] = '0';
    }

    sHash[nDifficulty] = '\0';

    mineblock<<<2,2>>>(nDifficulty, _nNonce, sHash)
    hipDeviceSynchronize();
    
    cout << "Block mined: " << sHash << endl;
}

inline string Block::_CalculateHash() const
{
    stringstream ss;
    ss << _nIndex << sPrevHash << _tTime << _sData << _nNonce;

    return sha256(ss.str());
}