#include "hip/hip_runtime.h"
//
// Created by Dave Nash on 20/10/2017.
//
#include <algorithm>
#include <cstdio>
#include <ctime>
#include <cstring>
#include "Block.cuh"
#include "sha256.cuh"
#include "sha256host.cuh"

Block::Block(uint32_t nIndexIn, const std::string &sDataIn) : _nIndex(nIndexIn), _sData(sDataIn)
{
    _nNonce = 0;
    _tTime = time(nullptr);

    sHash = _CalculateHash();
}

__device__ int strcmp_cuda(const char *str1, const char *str2)
{
    while (*str1 && (*str1 == *str2))
    {
        str1++;
        str2++;
    }
    return *(unsigned char *)str1 - *(unsigned char *)str2;
}

__device__ char* uint32_to_string(uint32_t num, char* buffer)
{
    int i = 0;
    do
    {
        buffer[i++] = '0' + (num % 10);
        num /= 10;
    } while (num > 0);
    buffer[i] = '\0';
    
    // Reverse the string
    for (int j = 0; j < i / 2; j++)
    {
        char temp = buffer[j];
        buffer[j] = buffer[i - j - 1];
        buffer[i - j - 1] = temp;
    }
    return buffer;
}

__device__ char* int64_to_string(int64_t num, char* buffer)
{
    int i = 0;
    bool isNegative = num < 0;
    if (isNegative) num = -num;

    do
    {
        buffer[i++] = '0' + (num % 10);
        num /= 10;
    } while (num > 0);
    
    if (isNegative) buffer[i++] = '-';
    buffer[i] = '\0';

    for (int j = 0; j < i / 2; j++)
    {
        char temp = buffer[j];
        buffer[j] = buffer[i - j - 1];
        buffer[i - j - 1] = temp;
    }
    return buffer;
}

__device__ void strcpy_cuda(char* dest, const char* src)
{
    while ((*dest++ = *src++) != '\0');
}

__device__ void strcat_cuda(char* dest, const char* src)
{
    while (*dest) dest++;
    while ((*dest++ = *src++) != '\0');
}

__global__ void mineBlockKernel(Block *b, char *str, uint32_t nDifficulty, char *resp)
{
    b->_nNonce++;
    char *sHash = b->_CalculateHashCuda();

    for (int i = 0; i < nDifficulty; i++)
    {
        resp[i] = sHash[i];
    }
    resp[nDifficulty] = '\0';

    if (strcmp_cuda(resp, str) == 0)
    {
        printf("Block mined: %s\n", sHash);
    }
}

void Block::MineBlock(uint32_t nDifficulty)
{
    Block *b = this;
    char cstr[nDifficulty + 1];
    for (uint32_t i = 0; i < nDifficulty; ++i)
    {
        cstr[i] = '0';
    }
    cstr[nDifficulty] = '\0';

    char *d_str;
    char *d_resp;
    hipMalloc(&d_str, sizeof(cstr));
    hipMalloc(&d_resp, nDifficulty + 1);
    hipMemcpy(d_str, cstr, sizeof(cstr), hipMemcpyHostToDevice);

    mineBlockKernel<<<1, 1>>>(b, d_str, nDifficulty, d_resp);

    hipFree(d_str);
    hipFree(d_resp);
}

__device__ inline char* Block::_CalculateHashCuda()
{
    char result[1024]; // Supondo que o tamanho máximo do hash seja 1024 caracteres

    char _nIndex_str[12];
    uint32_to_string(_nIndex, _nIndex_str);

    char _tTime_str[20];
    int64_to_string(_tTime, _tTime_str);

    char _nNonce_str[12];
    uint32_to_string(_nNonce, _nNonce_str);

    strcpy_cuda(result, _nIndex_str);
    strcat_cuda(result, _tTime_str);
    strcat_cuda(result, _nNonce_str);

    return SHA256CUDA::sha256(result);
}

inline std::string Block::_CalculateHash()
{
    std::stringstream ss;
    ss << _nIndex << sPrevHash << _tTime << _sData << _nNonce;

    return sha256(ss.str());
}