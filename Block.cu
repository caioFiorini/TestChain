#include "hip/hip_runtime.h"
//
// Created by Dave Nash on 20/10/2017.
//

#include "Block.cuh"
#include "sha256.cuh"
#include "string.h"

__global__ void calculateHashKernel(char* sHash)
{
    // Chame a função _CalculateHash() aqui e armazene o resultado em sHash
    Block* b;
    sHash = b._CalculateHash();
}

Block::Block(uint32_t nIndexIn, const string &sDataIn) : _nIndex(nIndexIn), _sData(sDataIn)
{
    _nNonce = 0;
    _tTime = time(nullptr);

    char* d_sHash;
    hipMalloc(&d_sHash, strlen(sHash));

    calculateHashKernel<<<1,1>>>(d_sHash)

    hipMemcpy(sHash, d_sHash, strlen(sHash), hipMemcpyDeviceToHost);

    hipFree(d_sHash);
}

__device__ void Block::mineBlock(char *str, uint32_t nDifficulty)
{
    (*_nNonce)++;
    sHash = _CalculateHash(); 

    if (sHash.substr(0, nDifficulty) != str) // std::string::substr não é suportado em CUDA
    {
        printf("Block mined: %s\n", sHash);
    }
}

__global__ void kernelMineBlock(char *d_str, uint32_t nDifficulty)
{
    mineBlock(d_nonce, d_hash, d_str, nDifficulty);
}

__host__ void Block::MineBlock(uint32_t nDifficulty)
{
    char cstr[nDifficulty + 1];
    for (uint32_t i = 0; i < nDifficulty; ++i)
    {
        cstr[i] = '0';
    }
    cstr[nDifficulty] = '\0';

    char *d_str;
    hipMalloc(&d_str, sizeof(cstr));
    hipMemcpy(d_str, cstr, sizeof(cstr), hipMemcpyHostToDevice);

    kernelMineBlock<<<1, 1>>>(d_str, nDifficulty);

    hipFree(d_str);
}

__device__ void Block::concatenate(char* result)
{
    char _nIndex_str[12]; 
    sprintf(_nIndex_str, "%u", _nIndex);

    char _tTime_str[20];
    sprintf(_tTime_str, "%lld", (long long) _tTime);

    char _nNonce_str[12];
    sprintf(_nNonce_str, "%u", _nNonce);

    strcpy(result, _nIndex_str);
    strcat(result, _tTime_str);
    strcat(result, _nNonce_str);
}

__device__ inline char* Block::_CalculateHash()
{
    char* result;
    concatenate(result)

    return sha256(result);
}
