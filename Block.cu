#include "Block.cuh"
#include "sha256.h"

Block::Block(uint32_t nIndexIn, const string &sDataIn) : _nIndex(nIndexIn), _sData(sDataIn)
{
    _nNonce = 0;
    _tTime = time(nullptr);

    sHash = _CalculateHash();
}

__device__ void Block::MineBlock(uint32_t nDifficulty)
{
    char cstr[nDifficulty + 1];
    for (uint32_t i = 0; i < nDifficulty; ++i)
    {
        cstr[i] = '0';
    }
    cstr[nDifficulty] = '\0';

    string str(cstr);

    do
    {
        _nNonce++;
        sHash = _CalculateHash();
    }while (sHash.substr(0, nDifficulty) != str);

    cout << "Block mined: " << sHash << endl;
}

//posso paralelizar essa parte do código.
void Block::MineBlock(uint32_t nDifficulty)
{
    hipMallocManaged(&_nNonce, sizeof(uint32_t));
    // nDifficulty + 1 -> seria adicionar espaço para o /0
    hipMallocManaged(&sHash, (nDifficulty+1) * sizeof(char));

    *_nNonce = 0;
    for(uint32_t i = 0; i < nDifficulty; i++){
        sHash[i] = '0';
    }

    sHash[nDifficulty] = '\0';

    mineblock<<<2,2>>>(nDifficulty, _nNonce, sHash)
    hipDeviceSynchronize();
    
    cout << "Block mined: " << sHash << endl;
}

__device__ char* Block::_CalculateHash() const
{
    stringstream ss;
    ss << _nIndex << sPrevHash << _tTime << _sData << _nNonce;

    return sha256(ss.str());
}