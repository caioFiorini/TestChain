#include "hip/hip_runtime.h"
#include "Block.cuh"
#include "sha256.cuh"


__device__ bool _IsValidHash(const char* hash, uint32_t nDifficulty) const {
    for (uint32_t i = 0; i < nDifficulty; ++i) {
        if (hash[i] != '0') {
            return false;
        }
    }
    return true;
}

__global__ void mineblock(uint32_t nDifficulty, uint32_t* nNonce, char* sHash, uint32_t index, time_t tTime, const char* sPrevHash, const char* sData) {
    uint32_t nonce = blockIdx.x * blockDim.x + threadIdx.x;

    char buffer_ss[1000];

    char hash[65];
    while (true) {
        // Calculate hash
        sprintf(buffer_ss, "%d%s%ld%s%d", index, sPrevHash, tTime, sData, nonce);
        
        sha256(buffer_ss, hash);

        for (int i = 0; i < 64; i++)
        {
            sHash[i] = hash[i];
        }

        hash[64] = '\0';

        if (_IsValidHash(hash, nDifficulty)) {
            strncpy(sHash, hash, 64);
            *nNonce = nonce;
            return;
        }
        nonce += gridDim.x * blockDim.x;
    }
}

Block::Block(uint32_t nIndexIn, const string &sDataIn) : _nIndex(nIndexIn), _sData(sDataIn) {
    hipMallocManaged(&_nNonce, sizeof(uint32_t));
    *_nNonce = 0;
    _tTime = time(nullptr);

    sHash = _CalculateHash();
}

void Block::MineBlock(uint32_t nDifficulty) {
    char* d_sHash;
    hipMallocManaged(&d_sHash, 65 * sizeof(char));
    
    char* d_sPrevHash;
    hipMallocManaged(&d_sPrevHash, sPrevHash.size() + 1);
    strcpy(d_sPrevHash, sPrevHash.c_str());

    char* d_sData;
    hipMallocManaged(&d_sData, _sData.size() + 1);
    strcpy(d_sData, _sData.c_str());

    mineblock<<<2, 2>>>(nDifficulty, _nNonce, d_sHash, _nIndex, _tTime, d_sPrevHash, d_sData);
    hipDeviceSynchronize();

    sHash = string(d_sHash);

    hipFree(d_sHash);
    hipFree(d_sPrevHash);
    hipFree(d_sData);

    cout << "Block mined: " << sHash << endl;
}

__device__ char* Block::_CalculateHash() const {
    stringstream ss;
    ss << _nIndex << sPrevHash << _tTime << _sData << *_nNonce;
    string hashString = sha256(ss.str());
    char* hash = new char[65];
    strncpy(hash, hashString.c_str(), 64);
    hash[64] = '\0';
    return hash;
}
